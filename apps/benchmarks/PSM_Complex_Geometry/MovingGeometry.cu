#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file ObjectRotatorGPU.cu
//! \author Philipp Suffa <philipp.suffa@fau.de>
//
//======================================================================================================================
#include "MovingGeometry.h"

#define SUB(dest,v1,v2) \
         dest.x=v1.x-v2.x; \
         dest.y=v1.y-v2.y; \
         dest.z=v1.z-v2.z;

#define ADD(dest,v1,v2) \
         dest.x=v1.x+v2.x; \
         dest.y=v1.y+v2.y; \
         dest.z=v1.z+v2.z;

#define ADDS1(dest,v1,s1) \
         dest.x=v1.x+s1; \
         dest.y=v1.y+s1; \
         dest.z=v1.z+s1;

namespace walberla
{

__global__ void resetFractionFieldGPUKernel( real_t * RESTRICT const fractionFieldData, int3 fieldSize, int3 stride) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < fieldSize.x  && y < fieldSize.y  && z < fieldSize.z )
   {
      const int idx = (x) + (y) * stride.y + (z) * stride.z;

      fractionFieldData[idx] = 0;
   }
}

void MovingGeometry::resetFractionField() {
   for (auto& block : *blocks_) {
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_FractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);

      int3 size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 stride_frac_field = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      dim3 _block(uint64_c(((16 < size.x - 2) ? 16 : size.x - 2)), uint64_c(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))), uint64_c(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))));
      dim3 _grid(uint64_c(( (size.x - 2) % (((16 < size.x - 2) ? 16 : size.x - 2)) == 0 ? (int64_t)(size.x - 2) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)) : ( (int64_t)(size.x - 2) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)) ) +1 )), uint64_c(( (size.y - 2) % (((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) == 0 ? (int64_t)(size.y - 2) / (int64_t)(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) : ( (int64_t)(size.y - 2) / (int64_t)(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) ) +1 )), uint64_c(( (size.z - 2) % (((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) == 0 ? (int64_t)(size.z - 2) / (int64_t)(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) : ( (int64_t)(size.z - 2) / (int64_t)(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) ) +1 )));

      resetFractionFieldGPUKernel<<<_grid, _block>>>(_data_FractionFieldGPU, size, stride_frac_field);
   }
}

__global__ void getFractionFieldFromGeometryMeshKernel(real_t * RESTRICT const _data_fractionFieldGPU, geoSize * RESTRICT const _data_geometryFieldGPU, int3 field_size, int3 field_stride, int3 geometry_field_size, int3 geometry_field_stride, double3 blockAABBMin, double3 meshAABBMin, double dx, int superSamplingDepth, int interpolationArea, double oneOverInterpolArea, double dxSS, double3 meshCenter, double3 rotationMatrixX, double3 rotationMatrixY, double3 rotationMatrixZ, double3 translation) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < field_size.x  && y < field_size.y  && z < field_size.z ) {
      const int idx = (x) + (y) * field_stride.y + (z) * field_stride.z;

      double dxHalf = 0.5 * dx;
      double3 cellCenter = { blockAABBMin.x + double(x) * dx + dxHalf, blockAABBMin.y + double(y) * dx + dxHalf, blockAABBMin.z + double(z) * dx + dxHalf };
      double3 rotatedCellCenter;
      double fraction = 0.0;

      //translation
      SUB(cellCenter, cellCenter, translation)

      //rotation
      SUB(cellCenter, cellCenter, meshCenter)
      rotatedCellCenter.x = cellCenter.x * rotationMatrixX.x + cellCenter.y * rotationMatrixX.y + cellCenter.z * rotationMatrixX.z;
      rotatedCellCenter.y = cellCenter.x * rotationMatrixY.x + cellCenter.y * rotationMatrixY.y + cellCenter.z * rotationMatrixY.z;
      rotatedCellCenter.z = cellCenter.x * rotationMatrixZ.x + cellCenter.y * rotationMatrixZ.y + cellCenter.z * rotationMatrixZ.z;
      ADD(rotatedCellCenter, rotatedCellCenter, meshCenter)

      double3 pointInGeometrySpace;
      SUB(pointInGeometrySpace, rotatedCellCenter, meshAABBMin);

      //get cell of geometry field
      int3 cellInGeometrySpace;
      cellInGeometrySpace.x = int(round((pointInGeometrySpace.x - dxHalf) / dxSS));
      cellInGeometrySpace.y = int(round((pointInGeometrySpace.y - dxHalf) / dxSS));
      cellInGeometrySpace.z = int(round((pointInGeometrySpace.z - dxHalf) / dxSS));

      if (cellInGeometrySpace.x < 0 || cellInGeometrySpace.x >= geometry_field_size.x ||
          cellInGeometrySpace.y < 0 || cellInGeometrySpace.y >= geometry_field_size.y ||
          cellInGeometrySpace.z < 0 || cellInGeometrySpace.z >= geometry_field_size.z )
      {
         fraction = 0.0;
      }
      else if (interpolationArea == 1){
         const int idx_geo = cellInGeometrySpace.x + cellInGeometrySpace.y * geometry_field_stride.y + cellInGeometrySpace.z * geometry_field_stride.z;
         fraction = _data_geometryFieldGPU[idx_geo];
      }
      else {
         double3 cellCenterInGeometrySpace = {double(cellInGeometrySpace.x) * dxSS + 0.5 * dxSS, double(cellInGeometrySpace.y) * dxSS + 0.5 * dxSS, double(cellInGeometrySpace.z) * dxSS + 0.5 * dxSS};
         double3 distanceToCellCenter;
         SUB(distanceToCellCenter, pointInGeometrySpace, cellCenterInGeometrySpace)
         int3 offset = {int(distanceToCellCenter.x / abs(distanceToCellCenter.x)), int(distanceToCellCenter.y / abs(distanceToCellCenter.y)), int(distanceToCellCenter.z / abs(distanceToCellCenter.z))};

         int3 iterationStart = {((offset.x < 0) ? -1 : 0) - superSamplingDepth - 1, ((offset.y < 0) ? -1 : 0) - superSamplingDepth - 1, ((offset.z < 0) ? -1 : 0) - superSamplingDepth - 1};
         int3 iterationEnd;
         ADDS1(iterationEnd, iterationStart, interpolationArea)

         for (int z = iterationStart.z; z < iterationEnd.z; ++z) {
            for (int y = iterationStart.y; y < iterationEnd.y; ++y) {
               for (int x = iterationStart.x; x < iterationEnd.x; ++x) {
                  int idx_geo = (cellInGeometrySpace.x + x) + (cellInGeometrySpace.y + y) * geometry_field_stride.y + (cellInGeometrySpace.z + z) * geometry_field_stride.z;
                  fraction += _data_geometryFieldGPU[idx_geo];
               }
            }
         }
         fraction *= oneOverInterpolArea;
      }
      _data_fractionFieldGPU[idx] = min(1.0, _data_fractionFieldGPU[idx] + fraction);
   }
}

void MovingGeometry::getFractionFieldFromGeometryMesh(uint_t timestep)  {

   Matrix3< real_t > rotationMat(rotationAxis_, real_t(timestep) * -rotationAngle_);
   double3 rotationMatrixX = {rotationMat[0], rotationMat[1], rotationMat[2]};
   double3 rotationMatrixY = {rotationMat[3], rotationMat[4], rotationMat[5]};
   double3 rotationMatrixZ = {rotationMat[6], rotationMat[7], rotationMat[8]};
   double3 translation = {translation_[0] * real_t(timestep), translation_[1] * real_t(timestep), translation_[2] * real_t(timestep)};

   for (auto& block : *blocks_) {
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_fractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);

      auto geometryFieldGPU = block.getData< gpu::GPUField<geoSize> >(geometryFieldGPUId_);
      geoSize * RESTRICT const _data_geometryFieldGPU = geometryFieldGPU->dataAt(0, 0, 0, 0);

      auto level         = blocks_->getLevel(block);
      auto dx     = double(blocks_->dx(level));
      double3 meshCenterGPU = {meshCenter[0], meshCenter[1], meshCenter[2]};
      auto blockAABB = block.getAABB();
      double3 blockAABBmin = {blockAABB.minCorner()[0], blockAABB.minCorner()[1], blockAABB.minCorner()[2]};
      double3 meshAABBmin = {meshAABB_.minCorner()[0], meshAABB_.minCorner()[1], meshAABB_.minCorner()[2]};

      uint_t interpolationArea = uint_t(pow(2, real_t(superSamplingDepth_)));
      real_t oneOverInterpolArea = 1.0 / pow(real_t(interpolationArea), 3);
      real_t dxSS = dx / pow(2, real_t(superSamplingDepth_));

      int3 field_size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 field_stride = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      int3 geometry_field_size = {int(geometryFieldGPU->xSize()), int(geometryFieldGPU->ySize()), int(geometryFieldGPU->zSize()) };
      int3 geometry_field_stride = {int(geometryFieldGPU->xStride()), int(geometryFieldGPU->yStride()), int(geometryFieldGPU->zStride())};

      dim3 _block(uint64_c(((16 < field_size.x - 2) ? 16 : field_size.x - 2)), uint64_c(((1024 < ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))) ? 1024 : ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))), uint64_c(((64 < ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))))) ? 64 : ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))))));
      dim3 _grid(uint64_c(( (field_size.x - 2) % (((16 < field_size.x - 2) ? 16 : field_size.x - 2)) == 0 ? (int64_t)(field_size.x - 2) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)) : ( (int64_t)(field_size.x - 2) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)) ) +1 )), uint64_c(( (field_size.y - 2) % (((1024 < ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))) ? 1024 : ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))) == 0 ? (int64_t)(field_size.y - 2) / (int64_t)(((1024 < ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))) ? 1024 : ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))) : ( (int64_t)(field_size.y - 2) / (int64_t)(((1024 < ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))) ? 1024 : ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))) ) +1 )), uint64_c(( (field_size.z - 2) % (((64 < ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))))) ? 64 : ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))))) == 0 ? (int64_t)(field_size.z - 2) / (int64_t)(((64 < ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))))) ? 64 : ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))))) : ( (int64_t)(field_size.z - 2) / (int64_t)(((64 < ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))))) ? 64 : ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))))) ) +1 )));

      getFractionFieldFromGeometryMeshKernel<<<_grid, _block>>>(_data_fractionFieldGPU, _data_geometryFieldGPU, field_size, field_stride,
                                                                      geometry_field_size, geometry_field_stride, blockAABBmin, meshAABBmin,
                                                                      dx, superSamplingDepth_, interpolationArea, oneOverInterpolArea, dxSS, meshCenterGPU,
                                                                      rotationMatrixX, rotationMatrixY, rotationMatrixZ, translation);
   }
}

__global__ void addStaticGeometryToFractionFieldKernel( real_t * RESTRICT const fractionFieldData, real_t * RESTRICT const staticFractionFieldData, int3 fieldSize, int3 stride) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < fieldSize.x  && y < fieldSize.y  && z < fieldSize.z )
   {
      const int idx = x + y * stride.y + z * stride.z;

      fractionFieldData[idx] = staticFractionFieldData[idx];
   }
}

void MovingGeometry::addStaticGeometryToFractionField() {
   for (auto& block : *blocks_) {
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_fractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);

      auto staticFractionFieldGPU = block.getData< gpu::GPUField<real_t> >(staticFractionFieldGPUId_);
      real_t * RESTRICT const _data_staticFractionFieldGPU = staticFractionFieldGPU->dataAt(0, 0, 0, 0);


      int3 size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 stride_frac_field = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      dim3 _block(uint64_c(((16 < size.x - 2) ? 16 : size.x - 2)), uint64_c(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))), uint64_c(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))));
      dim3 _grid(uint64_c(( (size.x - 2) % (((16 < size.x - 2) ? 16 : size.x - 2)) == 0 ? (int64_t)(size.x - 2) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)) : ( (int64_t)(size.x - 2) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)) ) +1 )), uint64_c(( (size.y - 2) % (((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) == 0 ? (int64_t)(size.y - 2) / (int64_t)(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) : ( (int64_t)(size.y - 2) / (int64_t)(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) ) +1 )), uint64_c(( (size.z - 2) % (((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) == 0 ? (int64_t)(size.z - 2) / (int64_t)(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) : ( (int64_t)(size.z - 2) / (int64_t)(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) ) +1 )));

      addStaticGeometryToFractionFieldKernel<<<_grid, _block>>>(_data_fractionFieldGPU, _data_staticFractionFieldGPU, size, stride_frac_field);
   }
}
} //namespace walberla
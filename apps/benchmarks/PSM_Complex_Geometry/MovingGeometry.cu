#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file ObjectRotatorGPU.cu
//! \author Philipp Suffa <philipp.suffa@fau.de>
//
//======================================================================================================================
#include "MovingGeometry.h"

#define SUB(dest,v1,v2) \
         dest.x=v1.x-v2.x; \
         dest.y=v1.y-v2.y; \
         dest.z=v1.z-v2.z;

#define ADD(dest,v1,v2) \
         dest.x=v1.x+v2.x; \
         dest.y=v1.y+v2.y; \
         dest.z=v1.z+v2.z;

#define ADDS1(dest,v1,s1) \
         dest.x=v1.x+s1; \
         dest.y=v1.y+s1; \
         dest.z=v1.z+s1;

namespace walberla
{

__global__ void resetFractionFieldGPUKernel( real_t * RESTRICT const fractionFieldData, int3 fieldSize, int3 stride) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < fieldSize.x  && y < fieldSize.y  && z < fieldSize.z )
   {
      const int idx = (x) + (y) * stride.y + (z) * stride.z;

      fractionFieldData[idx] = 0;
   }
}

void MovingGeometry::resetFractionField() {
   for (auto& block : *blocks_) {
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_FractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);

      int3 size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 stride_frac_field = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      dim3 _block(uint64_c(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)), uint64_c(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))), uint64_c(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))));
      dim3 _grid(uint64_c(( (size.x - 2 * ghostLayers_) % (((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) == 0 ? (int64_t)(size.x - 2 * ghostLayers_) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) : ( (int64_t)(size.x - 2 * ghostLayers_) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) ) +1 )), uint64_c(( (size.y - 2 * ghostLayers_) % (((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) == 0 ? (int64_t)(size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) : ( (int64_t)(size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) ) +1 )), uint64_c(( (size.z - 2 * ghostLayers_) % (((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) == 0 ? (int64_t)(size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) : ( (int64_t)(size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) ) +1 )));

      resetFractionFieldGPUKernel<<<_grid, _block>>>(_data_FractionFieldGPU, size, stride_frac_field);
   }
}

__global__ void getFractionFieldFromGeometryMeshKernel(real_t * RESTRICT const _data_fractionFieldGPU, geoSize * RESTRICT const _data_geometryFieldGPU, int3 field_size, int3 field_stride, int3 geometry_field_size, int3 geometry_field_stride, double3 blockAABBMin, double3 meshAABBMin, double dx, int superSamplingDepth, int interpolationStencilSize, double oneOverInterpolArea, double dxSS, double3 meshCenter, double3 rotationMatrixX, double3 rotationMatrixY, double3 rotationMatrixZ, double3 translation) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < field_size.x  && y < field_size.y  && z < field_size.z ) {
      const int idx = (x) + (y) * field_stride.y + (z) * field_stride.z;

      double dxHalf = 0.5 * dx;
      double3 vecDxSSHalf = {0.5 * dxSS, 0.5 * dxSS, 0.5 * dxSS};
      double3 cellCenter = { blockAABBMin.x + double(x) * dx + dxHalf, blockAABBMin.y + double(y) * dx + dxHalf, blockAABBMin.z + double(z) * dx + dxHalf };
      double3 rotatedCellCenter;

      //translation
      SUB(cellCenter, cellCenter, translation)

      //rotation
      SUB(cellCenter, cellCenter, meshCenter)
      rotatedCellCenter.x = cellCenter.x * rotationMatrixX.x + cellCenter.y * rotationMatrixX.y + cellCenter.z * rotationMatrixX.z;
      rotatedCellCenter.y = cellCenter.x * rotationMatrixY.x + cellCenter.y * rotationMatrixY.y + cellCenter.z * rotationMatrixY.z;
      rotatedCellCenter.z = cellCenter.x * rotationMatrixZ.x + cellCenter.y * rotationMatrixZ.y + cellCenter.z * rotationMatrixZ.z;
      ADD(rotatedCellCenter, rotatedCellCenter, meshCenter)

      double3 pointInGeometrySpace;
      SUB(pointInGeometrySpace, rotatedCellCenter, meshAABBMin);
      SUB(pointInGeometrySpace, pointInGeometrySpace, vecDxSSHalf);

      //get cell of geometry field
      int3 cellInGeometrySpace;
      cellInGeometrySpace.x = int((pointInGeometrySpace.x) / dxSS);
      cellInGeometrySpace.y = int((pointInGeometrySpace.y) / dxSS);
      cellInGeometrySpace.z = int((pointInGeometrySpace.z) / dxSS);

      double fraction = 0.0;

      if (cellInGeometrySpace.x < 0 || cellInGeometrySpace.x >= geometry_field_size.x ||
          cellInGeometrySpace.y < 0 || cellInGeometrySpace.y >= geometry_field_size.y ||
          cellInGeometrySpace.z < 0 || cellInGeometrySpace.z >= geometry_field_size.z )
      {
         fraction = 0.0;
      }
      else if (superSamplingDepth == 0){

         double3 cellCenterInGeometrySpace = {double(cellInGeometrySpace.x) * dxSS, double(cellInGeometrySpace.y) * dxSS, double(cellInGeometrySpace.z) * dxSS};
         double3 distanceToCellCenter;
         SUB(distanceToCellCenter, pointInGeometrySpace, cellCenterInGeometrySpace)
         int3 offset = {int(distanceToCellCenter.x / abs(distanceToCellCenter.x)), int(distanceToCellCenter.y / abs(distanceToCellCenter.y)), int(distanceToCellCenter.z / abs(distanceToCellCenter.z))};
         int3 iterationStart = {((offset.x < 0) ? -1 : 0), ((offset.y < 0) ? -1 : 0), ((offset.z < 0) ? -1 : 0)};
         int3 iterationEnd;
         ADDS1(iterationEnd, iterationStart, interpolationStencilSize)
         for (int zOff = iterationStart.z; zOff < iterationEnd.z; ++zOff) {
            for (int yOff = iterationStart.y; yOff < iterationEnd.y; ++yOff) {
               for (int xOff = iterationStart.x; xOff < iterationEnd.x; ++xOff) {
                  int idx_geo = (cellInGeometrySpace.x + xOff) + (cellInGeometrySpace.y + yOff) * geometry_field_stride.y + (cellInGeometrySpace.z + zOff) * geometry_field_stride.z;
                  fraction += _data_geometryFieldGPU[idx_geo];
               }
            }
         }
         fraction *= oneOverInterpolArea;
      }
      else {
         int halfInterpolationStencilSize = int(real_t(interpolationStencilSize) * 0.5);
         for (int zOff = -halfInterpolationStencilSize; zOff <= halfInterpolationStencilSize; ++zOff) {
            for (int yOff = -halfInterpolationStencilSize; yOff <= halfInterpolationStencilSize; ++yOff) {
               for (int xOff = -halfInterpolationStencilSize; xOff <= halfInterpolationStencilSize; ++xOff) {
                  int idx_geo = (cellInGeometrySpace.x + xOff) + (cellInGeometrySpace.y + yOff) * geometry_field_stride.y + (cellInGeometrySpace.z + zOff) * geometry_field_stride.z;
                  fraction += _data_geometryFieldGPU[idx_geo];
               }
            }
         }
         fraction *= oneOverInterpolArea;
      }
      _data_fractionFieldGPU[idx] = min(1.0, _data_fractionFieldGPU[idx] + fraction);
   }
}

void MovingGeometry::getFractionFieldFromGeometryMesh(uint_t timestep)  {

   Matrix3< real_t > rotationMat(rotationAxis_, real_t(timestep) * -rotationAngle_);
   double3 rotationMatrixX = {rotationMat[0], rotationMat[1], rotationMat[2]};
   double3 rotationMatrixY = {rotationMat[3], rotationMat[4], rotationMat[5]};
   double3 rotationMatrixZ = {rotationMat[6], rotationMat[7], rotationMat[8]};
   double3 translation = {translation_[0] * real_t(timestep), translation_[1] * real_t(timestep), translation_[2] * real_t(timestep)};

   for (auto& block : *blocks_) {
      uint_t level = blocks_->getLevel(block);
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_fractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);
      geoSize * RESTRICT const _data_geometryFieldGPU = geometryFieldGPU_->dataAt(0, 0, 0, 0);

      auto dx     = double(blocks_->dx(level));
      double3 meshCenterGPU = {meshCenter[0], meshCenter[1], meshCenter[2]};
      auto blockAABB = block.getAABB();
      double3 blockAABBmin = {blockAABB.minCorner()[0], blockAABB.minCorner()[1], blockAABB.minCorner()[2]};
      double3 meshAABBmin = {meshAABB_.minCorner()[0], meshAABB_.minCorner()[1], meshAABB_.minCorner()[2]};

      uint_t interpolationStencilSize = uint_t( pow(2, real_t(superSamplingDepth_)) + 1);
      auto oneOverInterpolArea = 1.0 / real_t( interpolationStencilSize * interpolationStencilSize * interpolationStencilSize);
      real_t dxSS = maxRefinementLevelPair_.second / pow(2, real_t(superSamplingDepth_));

      int3 field_size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 field_stride = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      int3 geometry_field_size = {int(geometryFieldGPU_->xSize()), int(geometryFieldGPU_->ySize()), int(geometryFieldGPU_->zSize()) };
      int3 geometry_field_stride = {int(geometryFieldGPU_->xStride()), int(geometryFieldGPU_->yStride()), int(geometryFieldGPU_->zStride())};

      dim3 _block(uint64_c(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)), uint64_c(((1024 < ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))) ? 1024 : ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))), uint64_c(((64 < ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))))) ? 64 : ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))))));
      dim3 _grid(uint64_c(( (field_size.x - 2 * ghostLayers_) % (((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)) == 0 ? (int64_t)(field_size.x - 2 * ghostLayers_) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)) : ( (int64_t)(field_size.x - 2 * ghostLayers_) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)) ) +1 )), uint64_c(( (field_size.y - 2 * ghostLayers_) % (((1024 < ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))) ? 1024 : ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))) == 0 ? (int64_t)(field_size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))) ? 1024 : ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))) : ( (int64_t)(field_size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))) ? 1024 : ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))) ) +1 )), uint64_c(( (field_size.z - 2 * ghostLayers_) % (((64 < ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))))) ? 64 : ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))))) == 0 ? (int64_t)(field_size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))))) ? 64 : ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))))) : ( (int64_t)(field_size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))))) ? 64 : ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))))) ) +1 )));

      getFractionFieldFromGeometryMeshKernel<<<_grid, _block>>>(_data_fractionFieldGPU, _data_geometryFieldGPU, field_size, field_stride,
                                                                      geometry_field_size, geometry_field_stride, blockAABBmin, meshAABBmin,
                                                                      dx, superSamplingDepth_, interpolationStencilSize, oneOverInterpolArea, dxSS, meshCenterGPU,
                                                                      rotationMatrixX, rotationMatrixY, rotationMatrixZ, translation);
   }
}

__global__ void addStaticGeometryToFractionFieldKernel( real_t * RESTRICT const fractionFieldData, real_t * RESTRICT const staticFractionFieldData, int3 fieldSize, int3 stride) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < fieldSize.x  && y < fieldSize.y  && z < fieldSize.z )
   {
      const int idx = x + y * stride.y + z * stride.z;

      fractionFieldData[idx] = min(1.0, fractionFieldData[idx] + staticFractionFieldData[idx]);
   }
}

void MovingGeometry::addStaticGeometryToFractionField() {
   for (auto& block : *blocks_) {
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_fractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);

      auto staticFractionFieldGPU = block.getData< gpu::GPUField<real_t> >(staticFractionFieldGPUId_);
      real_t * RESTRICT const _data_staticFractionFieldGPU = staticFractionFieldGPU->dataAt(0, 0, 0, 0);


      int3 size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 stride_frac_field = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      dim3 _block(uint64_c(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)), uint64_c(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))), uint64_c(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))));
      dim3 _grid(uint64_c(( (size.x - 2 * ghostLayers_) % (((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) == 0 ? (int64_t)(size.x - 2 * ghostLayers_) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) : ( (int64_t)(size.x - 2 * ghostLayers_) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) ) +1 )), uint64_c(( (size.y - 2 * ghostLayers_) % (((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) == 0 ? (int64_t)(size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) : ( (int64_t)(size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) ) +1 )), uint64_c(( (size.z - 2 * ghostLayers_) % (((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) == 0 ? (int64_t)(size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) : ( (int64_t)(size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) ) +1 )));

      addStaticGeometryToFractionFieldKernel<<<_grid, _block>>>(_data_fractionFieldGPU, _data_staticFractionFieldGPU, size, stride_frac_field);
   }
}
} //namespace walberla
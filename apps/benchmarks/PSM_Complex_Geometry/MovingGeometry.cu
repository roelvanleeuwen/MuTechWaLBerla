#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file ObjectRotatorGPU.cu
//! \author Philipp Suffa <philipp.suffa@fau.de>
//
//======================================================================================================================
#include "MovingGeometry.h"

#define SUB(dest,v1,v2) \
         dest.x=v1.x-v2.x; \
         dest.y=v1.y-v2.y; \
         dest.z=v1.z-v2.z;

#define ADD(dest,v1,v2) \
         dest.x=v1.x+v2.x; \
         dest.y=v1.y+v2.y; \
         dest.z=v1.z+v2.z;

namespace walberla
{

__global__ void resetFractionFieldGPUKernel( real_t * RESTRICT const fractionFieldData, int3 fieldSize, int3 stride) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < fieldSize.x  && y < fieldSize.y  && z < fieldSize.z )
   {
      const int idx = (x) + (y) * stride.y + (z) * stride.z;

      fractionFieldData[idx] = 0;
   }
}

void MovingGeometry::resetFractionField() {
   for (auto& block : *blocks_) {
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_FractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);

      int3 size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 stride_frac_field = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      dim3 _block(uint64_c(((16 < size.x - 2) ? 16 : size.x - 2)), uint64_c(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))), uint64_c(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))));
      dim3 _grid(uint64_c(( (size.x - 2) % (((16 < size.x - 2) ? 16 : size.x - 2)) == 0 ? (int64_t)(size.x - 2) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)) : ( (int64_t)(size.x - 2) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)) ) +1 )), uint64_c(( (size.y - 2) % (((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) == 0 ? (int64_t)(size.y - 2) / (int64_t)(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) : ( (int64_t)(size.y - 2) / (int64_t)(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) ) +1 )), uint64_c(( (size.z - 2) % (((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) == 0 ? (int64_t)(size.z - 2) / (int64_t)(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) : ( (int64_t)(size.z - 2) / (int64_t)(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) ) +1 )));

      resetFractionFieldGPUKernel<<<_grid, _block>>>(_data_FractionFieldGPU, size, stride_frac_field);
   }
}

__global__ void getFractionFieldFromGeometryMeshKernel(real_t * RESTRICT const _data_fractionFieldGPU, real_t * RESTRICT const _data_geometryFieldGPU, int3 field_size, int3 field_stride, int3 geometry_field_size, int3 geometry_field_stride, double3 blockAABBMin, double3 meshAABBMin, double dx, double3 meshCenter, double3 rotationMatrixX, double3 rotationMatrixY, double3 rotationMatrixZ, double3 translation) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < field_size.x  && y < field_size.y  && z < field_size.z ) {
      const int idx = (x) + (y) * field_stride.y + (z) * field_stride.z;

      double dxHalf = 0.5 * dx;
      double3 cellCenter = { blockAABBMin.x + double(x) * dx + dxHalf, blockAABBMin.y + double(y) * dx + dxHalf, blockAABBMin.z + double(z) * dx + dxHalf };
      double3 rotatedCellCenter;
      double fraction;

      //translation
      SUB(cellCenter, cellCenter, translation)

      //rotation
      SUB(cellCenter, cellCenter, meshCenter)
      rotatedCellCenter.x = cellCenter.x * rotationMatrixX.x + cellCenter.y * rotationMatrixX.y + cellCenter.z * rotationMatrixX.z;
      rotatedCellCenter.y = cellCenter.x * rotationMatrixY.x + cellCenter.y * rotationMatrixY.y + cellCenter.z * rotationMatrixY.z;
      rotatedCellCenter.z = cellCenter.x * rotationMatrixZ.x + cellCenter.y * rotationMatrixZ.y + cellCenter.z * rotationMatrixZ.z;
      ADD(rotatedCellCenter, rotatedCellCenter, meshCenter)

      //get cell of geometry field
      int3 cellInGeometrySpace;
      cellInGeometrySpace.x = int(round((rotatedCellCenter.x - meshAABBMin.x - dxHalf) / dx));
      cellInGeometrySpace.y = int(round((rotatedCellCenter.y - meshAABBMin.y - dxHalf) / dx));
      cellInGeometrySpace.z = int(round((rotatedCellCenter.z - meshAABBMin.z - dxHalf) / dx));

      if (cellInGeometrySpace.x < 0 || cellInGeometrySpace.x >= geometry_field_size.x ||
          cellInGeometrySpace.y < 0 || cellInGeometrySpace.y >= geometry_field_size.y ||
          cellInGeometrySpace.z < 0 || cellInGeometrySpace.z >= geometry_field_size.z )
      {
         fraction = 0.0;
      }
      else {
         const int idx_geo = cellInGeometrySpace.x + cellInGeometrySpace.y * geometry_field_stride.y + cellInGeometrySpace.z * geometry_field_stride.z;
         fraction = _data_geometryFieldGPU[idx_geo];
      }
      _data_fractionFieldGPU[idx] = min(1.0, _data_fractionFieldGPU[idx] + fraction);
   }
}

void MovingGeometry::getFractionFieldFromGeometryMesh(uint_t timestep)  {

   Matrix3< real_t > rotationMat(rotationAxis_, real_t(timestep) * -rotationAngle_);
   double3 rotationMatrixX = {rotationMat[0], rotationMat[1], rotationMat[2]};
   double3 rotationMatrixY = {rotationMat[3], rotationMat[4], rotationMat[5]};
   double3 rotationMatrixZ = {rotationMat[6], rotationMat[7], rotationMat[8]};
   double3 translation = {translation_[0] * real_t(timestep), translation_[1] * real_t(timestep), translation_[2] * real_t(timestep)};

   for (auto& block : *blocks_) {
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_fractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);

      auto geometryFieldGPU = block.getData< gpu::GPUField<real_t> >(geometryFieldGPUId_);
      real_t * RESTRICT const _data_geometryFieldGPU = geometryFieldGPU->dataAt(0, 0, 0, 0);

      auto level         = blocks_->getLevel(block);
      auto dx     = double(blocks_->dx(level));
      double3 meshCenterGPU = {meshCenter[0], meshCenter[1], meshCenter[2]};
      auto blockAABB = block.getAABB();
      double3 blockAABBmin = {blockAABB.minCorner()[0], blockAABB.minCorner()[1], blockAABB.minCorner()[2]};
      double3 meshAABBmin = {meshAABB_.minCorner()[0], meshAABB_.minCorner()[1], meshAABB_.minCorner()[2]};

      int3 field_size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 field_stride = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      int3 geometry_field_size = {int(geometryFieldGPU->xSize()), int(geometryFieldGPU->ySize()), int(geometryFieldGPU->zSize()) };
      int3 geometry_field_stride = {int(geometryFieldGPU->xStride()), int(geometryFieldGPU->yStride()), int(geometryFieldGPU->zStride())};

      dim3 _block(uint64_c(((16 < field_size.x - 2) ? 16 : field_size.x - 2)), uint64_c(((1024 < ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))) ? 1024 : ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))), uint64_c(((64 < ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))))) ? 64 : ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))))));
      dim3 _grid(uint64_c(( (field_size.x - 2) % (((16 < field_size.x - 2) ? 16 : field_size.x - 2)) == 0 ? (int64_t)(field_size.x - 2) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)) : ( (int64_t)(field_size.x - 2) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)) ) +1 )), uint64_c(( (field_size.y - 2) % (((1024 < ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))) ? 1024 : ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))) == 0 ? (int64_t)(field_size.y - 2) / (int64_t)(((1024 < ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))) ? 1024 : ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))) : ( (int64_t)(field_size.y - 2) / (int64_t)(((1024 < ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))) ? 1024 : ((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))) ) +1 )), uint64_c(( (field_size.z - 2) % (((64 < ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))))) ? 64 : ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))))) == 0 ? (int64_t)(field_size.z - 2) / (int64_t)(((64 < ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))))) ? 64 : ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))))) : ( (int64_t)(field_size.z - 2) / (int64_t)(((64 < ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))))))) ? 64 : ((field_size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))) ? field_size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)*((field_size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2)))) ? field_size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2) ? 16 : field_size.x - 2))))))))) ) +1 )));

      getFractionFieldFromGeometryMeshKernel<<<_grid, _block>>>(_data_fractionFieldGPU, _data_geometryFieldGPU, field_size, field_stride, geometry_field_size, geometry_field_stride, blockAABBmin, meshAABBmin, dx, meshCenterGPU, rotationMatrixX, rotationMatrixY, rotationMatrixZ, translation);
   }
}

__global__ void addStaticGeometryToFractionFieldKernel( real_t * RESTRICT const fractionFieldData, real_t * RESTRICT const staticFractionFieldData, int3 fieldSize, int3 stride) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < fieldSize.x  && y < fieldSize.y  && z < fieldSize.z )
   {
      const int idx = x + y * stride.y + z * stride.z;

      fractionFieldData[idx] = staticFractionFieldData[idx];
   }
}

void MovingGeometry::addStaticGeometryToFractionField() {
   for (auto& block : *blocks_) {
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_fractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);

      auto staticFractionFieldGPU = block.getData< gpu::GPUField<real_t> >(staticFractionFieldGPUId_);
      real_t * RESTRICT const _data_staticFractionFieldGPU = staticFractionFieldGPU->dataAt(0, 0, 0, 0);


      int3 size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 stride_frac_field = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      dim3 _block(uint64_c(((16 < size.x - 2) ? 16 : size.x - 2)), uint64_c(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))), uint64_c(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))));
      dim3 _grid(uint64_c(( (size.x - 2) % (((16 < size.x - 2) ? 16 : size.x - 2)) == 0 ? (int64_t)(size.x - 2) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)) : ( (int64_t)(size.x - 2) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)) ) +1 )), uint64_c(( (size.y - 2) % (((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) == 0 ? (int64_t)(size.y - 2) / (int64_t)(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) : ( (int64_t)(size.y - 2) / (int64_t)(((1024 < ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))) ? 1024 : ((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))) ) +1 )), uint64_c(( (size.z - 2) % (((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) == 0 ? (int64_t)(size.z - 2) / (int64_t)(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) : ( (int64_t)(size.z - 2) / (int64_t)(((64 < ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))))))) ? 64 : ((size.z - 2 < ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))) ? size.z - 2 : ((int64_t)(256) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)*((size.y - 2 < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2)))) ? size.y - 2 : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2) ? 16 : size.x - 2))))))))) ) +1 )));

      addStaticGeometryToFractionFieldKernel<<<_grid, _block>>>(_data_fractionFieldGPU, _data_staticFractionFieldGPU, size, stride_frac_field);
   }
}
} //namespace walberla
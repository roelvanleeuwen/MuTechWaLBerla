#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file ObjectRotatorGPU.cu
//! \author Philipp Suffa <philipp.suffa@fau.de>
//
//======================================================================================================================
#include "MovingGeometry.h"

#define SUB(dest,v1,v2) \
         dest.x=v1.x-v2.x; \
         dest.y=v1.y-v2.y; \
         dest.z=v1.z-v2.z;

#define ADD(dest,v1,v2) \
         dest.x=v1.x+v2.x; \
         dest.y=v1.y+v2.y; \
         dest.z=v1.z+v2.z;

#define ADDS1(dest,v1,s1) \
         dest.x=v1.x+s1; \
         dest.y=v1.y+s1; \
         dest.z=v1.z+s1;

namespace walberla
{

__global__ void resetFractionFieldGPUKernel( real_t * RESTRICT const fractionFieldData, int3 fieldSize, int3 stride) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < fieldSize.x  && y < fieldSize.y  && z < fieldSize.z )
   {
      const int idx = (x) + (y) * stride.y + (z) * stride.z;

      fractionFieldData[idx] = 0;
   }
}

void MovingGeometry::resetFractionField() {
   for (auto& block : *blocks_) {
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_FractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);

      int3 size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 stride_frac_field = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      dim3 _block(uint64_c(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)), uint64_c(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))), uint64_c(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))));
      dim3 _grid(uint64_c(( (size.x - 2 * ghostLayers_) % (((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) == 0 ? (int64_t)(size.x - 2 * ghostLayers_) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) : ( (int64_t)(size.x - 2 * ghostLayers_) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) ) +1 )), uint64_c(( (size.y - 2 * ghostLayers_) % (((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) == 0 ? (int64_t)(size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) : ( (int64_t)(size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) ) +1 )), uint64_c(( (size.z - 2 * ghostLayers_) % (((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) == 0 ? (int64_t)(size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) : ( (int64_t)(size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) ) +1 )));

      resetFractionFieldGPUKernel<<<_grid, _block>>>(_data_FractionFieldGPU, size, stride_frac_field);
   }
}

__global__ void getFractionFieldFromGeometryMeshKernel(real_t * RESTRICT const _data_fractionFieldGPU, geoSize * RESTRICT const _data_geometryFieldGPU, int3 field_size, int3 field_stride, int3 geometry_field_size, int3 geometry_field_stride, double3 blockAABBMin, double3 meshAABBMin, float3 dxyz, int superSamplingDepth, int interpolationStencilSize, double oneOverInterpolArea, float3 dxyzSS, double3 meshCenter, double3 rotationMatrixX, double3 rotationMatrixY, double3 rotationMatrixZ, double3 translation) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < field_size.x  && y < field_size.y  && z < field_size.z ) {
      const int idx = (x) + (y) * field_stride.y + (z) * field_stride.z;

      double3 vecDxSSHalf = {0.5 * dxyzSS.x, 0.5 * dxyzSS.y, 0.5 * dxyzSS.z};
      double3 cellCenter = { blockAABBMin.x + double(x) * dxyz.x + 0.5 * dxyz.x, blockAABBMin.y + double(y) * dxyz.y +  0.5 * dxyz.y, blockAABBMin.z + double(z) * dxyz.z +  0.5 * dxyz.z };
      double3 rotatedCellCenter;

      //translation
      SUB(cellCenter, cellCenter, translation)

      //rotation
      SUB(cellCenter, cellCenter, meshCenter)
      rotatedCellCenter.x = cellCenter.x * rotationMatrixX.x + cellCenter.y * rotationMatrixX.y + cellCenter.z * rotationMatrixX.z;
      rotatedCellCenter.y = cellCenter.x * rotationMatrixY.x + cellCenter.y * rotationMatrixY.y + cellCenter.z * rotationMatrixY.z;
      rotatedCellCenter.z = cellCenter.x * rotationMatrixZ.x + cellCenter.y * rotationMatrixZ.y + cellCenter.z * rotationMatrixZ.z;
      ADD(rotatedCellCenter, rotatedCellCenter, meshCenter)

      double3 pointInGeometrySpace;
      SUB(pointInGeometrySpace, rotatedCellCenter, meshAABBMin);
      SUB(pointInGeometrySpace, pointInGeometrySpace, vecDxSSHalf);

      //get cell of geometry field
      int3 cellInGeometrySpace;
      cellInGeometrySpace.x = int((pointInGeometrySpace.x) / dxyzSS.x);
      cellInGeometrySpace.y = int((pointInGeometrySpace.y) / dxyzSS.y);
      cellInGeometrySpace.z = int((pointInGeometrySpace.z) / dxyzSS.z);

      double fraction = 0.0;

      if (cellInGeometrySpace.x < 0 || cellInGeometrySpace.x >= geometry_field_size.x ||
          cellInGeometrySpace.y < 0 || cellInGeometrySpace.y >= geometry_field_size.y ||
          cellInGeometrySpace.z < 0 || cellInGeometrySpace.z >= geometry_field_size.z )
      {
         fraction = 0.0;
      }
      else if (superSamplingDepth == 0){

         double3 cellCenterInGeometrySpace = {double(cellInGeometrySpace.x) * dxyzSS.x, double(cellInGeometrySpace.y) * dxyzSS.y, double(cellInGeometrySpace.z) * dxyzSS.z};
         double3 distanceToCellCenter;
         SUB(distanceToCellCenter, pointInGeometrySpace, cellCenterInGeometrySpace)
         int3 offset = {int(distanceToCellCenter.x / abs(distanceToCellCenter.x)), int(distanceToCellCenter.y / abs(distanceToCellCenter.y)), int(distanceToCellCenter.z / abs(distanceToCellCenter.z))};
         int3 iterationStart = {((offset.x < 0) ? -1 : 0), ((offset.y < 0) ? -1 : 0), ((offset.z < 0) ? -1 : 0)};
         int3 iterationEnd;
         ADDS1(iterationEnd, iterationStart, interpolationStencilSize)
         for (int zOff = iterationStart.z; zOff < iterationEnd.z; ++zOff) {
            for (int yOff = iterationStart.y; yOff < iterationEnd.y; ++yOff) {
               for (int xOff = iterationStart.x; xOff < iterationEnd.x; ++xOff) {
                  int idx_geo = (cellInGeometrySpace.x + xOff) + (cellInGeometrySpace.y + yOff) * geometry_field_stride.y + (cellInGeometrySpace.z + zOff) * geometry_field_stride.z;
                  fraction += _data_geometryFieldGPU[idx_geo];
               }
            }
         }
         fraction *= oneOverInterpolArea;
      }
      else {
         int halfInterpolationStencilSize = int(real_t(interpolationStencilSize) * 0.5);
         for (int zOff = -halfInterpolationStencilSize; zOff <= halfInterpolationStencilSize; ++zOff) {
            for (int yOff = -halfInterpolationStencilSize; yOff <= halfInterpolationStencilSize; ++yOff) {
               for (int xOff = -halfInterpolationStencilSize; xOff <= halfInterpolationStencilSize; ++xOff) {
                  int idx_geo = (cellInGeometrySpace.x + xOff) + (cellInGeometrySpace.y + yOff) * geometry_field_stride.y + (cellInGeometrySpace.z + zOff) * geometry_field_stride.z;
                  fraction += _data_geometryFieldGPU[idx_geo];
               }
            }
         }
         fraction *= oneOverInterpolArea;
      }
      _data_fractionFieldGPU[idx] = min(1.0, _data_fractionFieldGPU[idx] + fraction);
   }
}

void MovingGeometry::getFractionFieldFromGeometryMesh(uint_t timestep)  {

   auto geometryMovement = movementFunction_(timestep);
   Matrix3<real_t>rotationMat(geometryMovement.rotationAxis, -geometryMovement.rotationAngle);
   double3 rotationMatrixX = {rotationMat[0], rotationMat[1], rotationMat[2]};
   double3 rotationMatrixY = {rotationMat[3], rotationMat[4], rotationMat[5]};
   double3 rotationMatrixZ = {rotationMat[6], rotationMat[7], rotationMat[8]};
   double3 translation = {geometryMovement.translationVector[0], geometryMovement.translationVector[1], geometryMovement.translationVector[2]};

   for (auto& block : *blocks_) {
      if(!geometryMovement.movementBoundingBox.intersects(block.getAABB()) )
         continue;
      uint_t level = blocks_->getLevel(block);
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_fractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);
      geoSize * RESTRICT const _data_geometryFieldGPU = geometryFieldGPU_->dataAt(0, 0, 0, 0);

      float3 dxyz = {float(blocks_->dx(level)), float(blocks_->dy(level)), float(blocks_->dz(level))};
      double3 meshCenterGPU = {meshCenter[0], meshCenter[1], meshCenter[2]};
      auto blockAABB = block.getAABB();
      double3 blockAABBmin = {blockAABB.minCorner()[0], blockAABB.minCorner()[1], blockAABB.minCorner()[2]};
      double3 meshAABBmin = {meshAABB_.minCorner()[0], meshAABB_.minCorner()[1], meshAABB_.minCorner()[2]};

      uint_t interpolationStencilSize = uint_t( pow(2, real_t(superSamplingDepth_)) + 1);
      auto oneOverInterpolArea = 1.0 / real_t( interpolationStencilSize * interpolationStencilSize * interpolationStencilSize);
      Vector3<real_t> dxyzSSreal_t = maxRefinementDxyz_ / pow(2, real_t(superSamplingDepth_));
      float3 dxyzSS = {float(dxyzSSreal_t[0]), float(dxyzSSreal_t[1]), float(dxyzSSreal_t[2])};

      int3 field_size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 field_stride = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      int3 geometry_field_size = {int(geometryFieldGPU_->xSize()), int(geometryFieldGPU_->ySize()), int(geometryFieldGPU_->zSize()) };
      int3 geometry_field_stride = {int(geometryFieldGPU_->xStride()), int(geometryFieldGPU_->yStride()), int(geometryFieldGPU_->zStride())};

      dim3 _block(uint64_c(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)), uint64_c(((1024 < ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))) ? 1024 : ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))), uint64_c(((64 < ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))))) ? 64 : ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))))));
      dim3 _grid(uint64_c(( (field_size.x - 2 * ghostLayers_) % (((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)) == 0 ? (int64_t)(field_size.x - 2 * ghostLayers_) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)) : ( (int64_t)(field_size.x - 2 * ghostLayers_) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)) ) +1 )), uint64_c(( (field_size.y - 2 * ghostLayers_) % (((1024 < ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))) ? 1024 : ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))) == 0 ? (int64_t)(field_size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))) ? 1024 : ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))) : ( (int64_t)(field_size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))) ? 1024 : ((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))) ) +1 )), uint64_c(( (field_size.z - 2 * ghostLayers_) % (((64 < ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))))) ? 64 : ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))))) == 0 ? (int64_t)(field_size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))))) ? 64 : ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))))) : ( (int64_t)(field_size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))))))) ? 64 : ((field_size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))) ? field_size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)*((field_size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_)))) ? field_size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < field_size.x - 2 * ghostLayers_) ? 16 : field_size.x - 2 * ghostLayers_))))))))) ) +1 )));

      getFractionFieldFromGeometryMeshKernel<<<_grid, _block>>>(_data_fractionFieldGPU, _data_geometryFieldGPU, field_size, field_stride,
                                                                      geometry_field_size, geometry_field_stride, blockAABBmin, meshAABBmin,
                                                                      dxyz, superSamplingDepth_, interpolationStencilSize, oneOverInterpolArea, dxyzSS, meshCenterGPU,
                                                                      rotationMatrixX, rotationMatrixY, rotationMatrixZ, translation);
   }
}

__global__ void addStaticGeometryToFractionFieldKernel( real_t * RESTRICT const fractionFieldData, real_t * RESTRICT const staticFractionFieldData, int3 fieldSize, int3 stride) {
   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < fieldSize.x  && y < fieldSize.y  && z < fieldSize.z )
   {
      const int idx = x + y * stride.y + z * stride.z;

      fractionFieldData[idx] = min(1.0, fractionFieldData[idx] + staticFractionFieldData[idx]);
   }
}

void MovingGeometry::addStaticGeometryToFractionField() {
   for (auto& block : *blocks_) {
      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_fractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);

      auto staticFractionFieldGPU = block.getData< gpu::GPUField<real_t> >(staticFractionFieldGPUId_);
      real_t * RESTRICT const _data_staticFractionFieldGPU = staticFractionFieldGPU->dataAt(0, 0, 0, 0);


      int3 size = {int(fractionFieldGPU->xSizeWithGhostLayer()), int(fractionFieldGPU->ySizeWithGhostLayer()), int(fractionFieldGPU->zSizeWithGhostLayer()) };
      int3 stride_frac_field = {int(fractionFieldGPU->xStride()), int(fractionFieldGPU->yStride()), int(fractionFieldGPU->zStride())};

      dim3 _block(uint64_c(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)), uint64_c(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))), uint64_c(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))));
      dim3 _grid(uint64_c(( (size.x - 2 * ghostLayers_) % (((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) == 0 ? (int64_t)(size.x - 2 * ghostLayers_) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) : ( (int64_t)(size.x - 2 * ghostLayers_) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) ) +1 )), uint64_c(( (size.y - 2 * ghostLayers_) % (((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) == 0 ? (int64_t)(size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) : ( (int64_t)(size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) ) +1 )), uint64_c(( (size.z - 2 * ghostLayers_) % (((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) == 0 ? (int64_t)(size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) : ( (int64_t)(size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) ) +1 )));

      addStaticGeometryToFractionFieldKernel<<<_grid, _block>>>(_data_fractionFieldGPU, _data_staticFractionFieldGPU, size, stride_frac_field);
   }
}

__global__ void updateObjectVelocityFieldKernel(real_t * RESTRICT const _data_objectVelocityFieldGPU, real_t * RESTRICT const _data_fractionFieldGPU, int3 field_size, int3 field_stride, int fStride, double3 blockAABBMin, double3 dxyz, double3 meshCenter, double3 angularVel, double3 translationSpeed, bool timeDependentMovement, double3 movementBoundingBoxMin, double3 movementBoundingBoxMax) {

   const int64_t x = blockDim.x*blockIdx.x + threadIdx.x ;
   const int64_t y = blockDim.y*blockIdx.y + threadIdx.y ;
   const int64_t z = blockDim.z*blockIdx.z + threadIdx.z ;
   if (x < field_size.x  && y < field_size.y  && z < field_size.z )
   {
      const int idx = (x) + (y) *field_stride.y + (z) *field_stride.z;

      double3 cellCenter = { blockAABBMin.x + double(x) * dxyz.x + 0.5 * dxyz.x, blockAABBMin.y + double(y) * dxyz.y + 0.5 * dxyz.y,
                             blockAABBMin.z + double(z) * dxyz.z + 0.5 * dxyz.z };

      if(timeDependentMovement)
      {
         if (_data_fractionFieldGPU[idx] <= 0.0)
         {
            _data_objectVelocityFieldGPU[idx + 0 * fStride] = 0.0;
            _data_objectVelocityFieldGPU[idx + 1 * fStride] = 0.0;
            _data_objectVelocityFieldGPU[idx + 2 * fStride] = 0.0;
            return;
         }
      }
      else {
         if (cellCenter.x + 0.5*dxyz.x < movementBoundingBoxMin.x || cellCenter.y + 0.5*dxyz.y < movementBoundingBoxMin.y || cellCenter.z + 0.5*dxyz.z < movementBoundingBoxMin.z
             || cellCenter.x - 0.5*dxyz.x  > movementBoundingBoxMax.x || cellCenter.y - 0.5*dxyz.y > movementBoundingBoxMax.y || cellCenter.z - 0.5*dxyz.z > movementBoundingBoxMax.z)
            return;
      }

      double3 distance = { (cellCenter.x - meshCenter.x) / dxyz.x, (cellCenter.y - meshCenter.y) / dxyz.y,
                           (cellCenter.z - meshCenter.z) / dxyz.z };

      double velX = angularVel.y * distance.z - angularVel.z * distance.y;
      double velY = angularVel.z * distance.x - angularVel.x * distance.z;
      double velZ = angularVel.x * distance.y - angularVel.y * distance.x;

      _data_objectVelocityFieldGPU[idx + 0 * fStride] = velX + translationSpeed.x / dxyz.x;
      _data_objectVelocityFieldGPU[idx + 1 * fStride] = velY + translationSpeed.y / dxyz.y;
      _data_objectVelocityFieldGPU[idx + 2 * fStride] = velZ + translationSpeed.z / dxyz.z;
   }
}


void MovingGeometry::updateObjectVelocityField(uint_t timestep) {
   auto geometryMovement = movementFunction_(timestep+1);
   auto geometryMovementLastTimestep = movementFunction_(timestep);
   const Vector3<real_t> dxyz_root = Vector3<real_t>(blocks_->dx(0), blocks_->dy(0), blocks_->dz(0));
   geometryMovement.movementBoundingBox.extend(dxyz_root);

   //update object velocity field only on 0th timestep for time independent movement
   if(!geometryMovement.timeDependentMovement && timestep > 0)
      return;

   auto rotationSpeed = geometryMovement.rotationAngle - geometryMovementLastTimestep.rotationAngle;
   auto translationSpeed = geometryMovement.translationVector - geometryMovementLastTimestep.translationVector;
   double3 translationSpeedGPU = {translationSpeed[0], translationSpeed[1], translationSpeed[2]};
   double3 angularVel = {geometryMovement.rotationAxis[0] * rotationSpeed, geometryMovement.rotationAxis[1] * rotationSpeed, geometryMovement.rotationAxis[2] * rotationSpeed};
   double3 meshCenterGPU = {meshCenter[0], meshCenter[1], meshCenter[2]};
   double3 movementBoundingBoxMin = {geometryMovement.movementBoundingBox.xMin(), geometryMovement.movementBoundingBox.yMin(), geometryMovement.movementBoundingBox.zMin()};
   double3 movementBoundingBoxMax = {geometryMovement.movementBoundingBox.xMax(), geometryMovement.movementBoundingBox.yMax(), geometryMovement.movementBoundingBox.zMax()};
   for (auto& block : *blocks_)
   {
      if(!geometryMovement.movementBoundingBox.intersects(block.getAABB()) )
         continue;

      auto level = blocks_->getLevel(block);
      double3 dxyz = {double(blocks_->dx(level)), double(blocks_->dy(level)), double(blocks_->dz(level))};

      auto blockAABB = block.getAABB();
      double3 blockAABBMin = {blockAABB.minCorner()[0], blockAABB.minCorner()[1], blockAABB.minCorner()[2]};

      auto fractionFieldGPU = block.getData< gpu::GPUField<real_t> >(fractionFieldId_);
      real_t * RESTRICT const _data_fractionFieldGPU = fractionFieldGPU->dataAt(0, 0, 0, 0);

      auto objectVelocityFieldGPU = block.getData< gpu::GPUField<real_t> >(objectVelocityId_);
      real_t * RESTRICT const _data_objectVelocityFieldGPU = objectVelocityFieldGPU->dataAt(0, 0, 0, 0);


      int3 size = {int(objectVelocityFieldGPU->xSizeWithGhostLayer()), int(objectVelocityFieldGPU->ySizeWithGhostLayer()), int(objectVelocityFieldGPU->zSizeWithGhostLayer()) };
      int3 stride_frac_field = {int(objectVelocityFieldGPU->xStride()), int(objectVelocityFieldGPU->yStride()), int(objectVelocityFieldGPU->zStride())};
      int fStride = objectVelocityFieldGPU->fStride();

      dim3 _block(uint64_c(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)), uint64_c(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))), uint64_c(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))));
      dim3 _grid(uint64_c(( (size.x - 2 * ghostLayers_) % (((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) == 0 ? (int64_t)(size.x - 2 * ghostLayers_) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) : ( (int64_t)(size.x - 2 * ghostLayers_) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)) ) +1 )), uint64_c(( (size.y - 2 * ghostLayers_) % (((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) == 0 ? (int64_t)(size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) : ( (int64_t)(size.y - 2 * ghostLayers_) / (int64_t)(((1024 < ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))) ? 1024 : ((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))) ) +1 )), uint64_c(( (size.z - 2 * ghostLayers_) % (((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) == 0 ? (int64_t)(size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) : ( (int64_t)(size.z - 2 * ghostLayers_) / (int64_t)(((64 < ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))))))) ? 64 : ((size.z - 2 * ghostLayers_ < ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))) ? size.z - 2 * ghostLayers_ : ((int64_t)(256) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)*((size.y - 2 * ghostLayers_ < 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_)))) ? size.y - 2 * ghostLayers_ : 16*((int64_t)(16) / (int64_t)(((16 < size.x - 2 * ghostLayers_) ? 16 : size.x - 2 * ghostLayers_))))))))) ) +1 )));

      updateObjectVelocityFieldKernel<<<_grid, _block>>>(_data_objectVelocityFieldGPU, _data_fractionFieldGPU, size, stride_frac_field, fStride, blockAABBMin, dxyz, meshCenterGPU, angularVel, translationSpeedGPU, geometryMovement.timeDependentMovement, movementBoundingBoxMin, movementBoundingBoxMax);
   }
}

















} //namespace walberla
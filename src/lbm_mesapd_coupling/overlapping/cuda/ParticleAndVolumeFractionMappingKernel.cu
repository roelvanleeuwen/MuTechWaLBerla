#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file ParticleAndVolumeFractionMappingKernel.cu
//! \ingroup lbm_mesapd_coupling
//! \author Samuel Kemmler <samuel.kemmler@fau.de>
//
//======================================================================================================================

#include "lbm_mesapd_coupling/DataTypesGPU.h"

#include <assert.h>

#include "ParticleAndVolumeFractionMappingKernel.h"

namespace walberla
{
namespace lbm_mesapd_coupling
{
namespace psm
{
namespace cuda
{

__global__ void resetKernel(walberla::cuda::FieldAccessor< PSMCell_T > field)
{
   field.set(blockIdx, threadIdx);
   for (uint i = 0; i < MaxParticlesPerCell; i++)
   {
      field.get().overlapFractions[i] = 0.0;
      field.get().uids[i]             = id_t(0);
   }
   field.get().index = 0;
}

// TODO: look for better mapping method
__global__ void particleAndVolumeFractionMappingKernel(walberla::cuda::FieldAccessor< PSMCell_T > field,
                                                       double3 spherePosition, real_t sphereRadius, double3 blockStart,
                                                       double3 dx, int3 nSamples, id_t uid)
{
   field.set(blockIdx, threadIdx);
   double3 sampleDistance       = { 1.0 / (nSamples.x + 1) * dx.x, 1.0 / (nSamples.y + 1) * dx.y,
                                    1.0 / (nSamples.z + 1) * dx.z };
   double3 startSamplingPoint   = { (blockStart.x + threadIdx.x * dx.x + sampleDistance.x),
                                    (blockStart.y + blockIdx.x * dx.y + sampleDistance.y),
                                    (blockStart.z + blockIdx.y * dx.z + sampleDistance.z) };
   double3 currentSamplingPoint = startSamplingPoint;

   double3 minCornerSphere = { spherePosition.x - sphereRadius, spherePosition.y - sphereRadius,
                               spherePosition.z - sphereRadius };
   double3 maxCornerSphere = { spherePosition.x + sphereRadius, spherePosition.y + sphereRadius,
                               spherePosition.z + sphereRadius };

   if (startSamplingPoint.x + dx.x > minCornerSphere.x && startSamplingPoint.x < maxCornerSphere.x &&
       startSamplingPoint.y + dx.y > minCornerSphere.y && startSamplingPoint.y < maxCornerSphere.y &&
       startSamplingPoint.z + dx.z > minCornerSphere.z && startSamplingPoint.z < maxCornerSphere.z)
   {
      for (uint_t z = 0; z < nSamples.z; z++)
      {
         currentSamplingPoint.y = startSamplingPoint.y;
         for (uint_t y = 0; y < nSamples.y; y++)
         {
            currentSamplingPoint.x = startSamplingPoint.x;
            for (uint_t x = 0; x < nSamples.x; x++)
            {
               if ((currentSamplingPoint.x - spherePosition.x) * (currentSamplingPoint.x - spherePosition.x) +
                      (currentSamplingPoint.y - spherePosition.y) * (currentSamplingPoint.y - spherePosition.y) +
                      (currentSamplingPoint.z - spherePosition.z) * (currentSamplingPoint.z - spherePosition.z) <=
                   sphereRadius * sphereRadius)
               {
                  field.get().overlapFractions[field.get().index] += 1.0;
               }
               currentSamplingPoint.x += sampleDistance.x;
            }
            currentSamplingPoint.y += sampleDistance.y;
         }
         currentSamplingPoint.z += sampleDistance.z;
      }

      field.get().overlapFractions[field.get().index] *= 1.0 / (nSamples.x * nSamples.y * nSamples.z);
      if (field.get().overlapFractions[field.get().index] > 0)
      {
         field.get().uids[field.get().index] = uid;
         field.get().index += 1;
      }
      assert(field.get().index < 8);
   }
}

} // namespace cuda
} // namespace psm
} // namespace lbm_mesapd_coupling
} // namespace walberla

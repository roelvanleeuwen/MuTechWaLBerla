#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \file ParticleAndVolumeFractionMappingKernel.cu
//! \ingroup lbm_mesapd_coupling
//! \author Samuel Kemmler <samuel.kemmler@fau.de>
//
//======================================================================================================================

#include "lbm_mesapd_coupling/DataTypesGPU.h"

#include <assert.h>

#include "ParticleAndVolumeFractionMappingKernel.h"

namespace walberla
{
namespace lbm_mesapd_coupling
{
namespace psm
{
namespace cuda
{

/*__global__ void resetKernelAoS(walberla::cuda::FieldAccessor< ParticleAndVolumeFractionAoS_T > field)
{
   field.set(blockIdx, threadIdx);
   for (uint i = 0; i < MaxParticlesPerCell; i++)
   {
      field.get().overlapFractions[i] = 0.0;
      field.get().uids[i]             = id_t(0);
   }
   field.get().index = 0;
}

// TODO: look for better mapping method
__global__ void
   particleAndVolumeFractionMappingKernelAoS(walberla::cuda::FieldAccessor< ParticleAndVolumeFractionAoS_T > field,
                                             double3 spherePosition, real_t sphereRadius, double3 blockStart,
                                             double3 dx, int3 nSamples, id_t uid)
{
   field.set(blockIdx, threadIdx);
   double3 sampleDistance       = { 1.0 / (nSamples.x + 1) * dx.x, 1.0 / (nSamples.y + 1) * dx.y,
                                    1.0 / (nSamples.z + 1) * dx.z };
   double3 startSamplingPoint   = { (blockStart.x + threadIdx.x * dx.x + sampleDistance.x),
                                    (blockStart.y + blockIdx.x * dx.y + sampleDistance.y),
                                    (blockStart.z + blockIdx.y * dx.z + sampleDistance.z) };
   double3 currentSamplingPoint = startSamplingPoint;

   double3 minCornerSphere = { spherePosition.x - sphereRadius, spherePosition.y - sphereRadius,
                               spherePosition.z - sphereRadius };
   double3 maxCornerSphere = { spherePosition.x + sphereRadius, spherePosition.y + sphereRadius,
                               spherePosition.z + sphereRadius };

   if (startSamplingPoint.x + dx.x > minCornerSphere.x && startSamplingPoint.x < maxCornerSphere.x &&
       startSamplingPoint.y + dx.y > minCornerSphere.y && startSamplingPoint.y < maxCornerSphere.y &&
       startSamplingPoint.z + dx.z > minCornerSphere.z && startSamplingPoint.z < maxCornerSphere.z)
   {
      for (uint_t z = 0; z < nSamples.z; z++)
      {
         currentSamplingPoint.y = startSamplingPoint.y;
         for (uint_t y = 0; y < nSamples.y; y++)
         {
            currentSamplingPoint.x = startSamplingPoint.x;
            for (uint_t x = 0; x < nSamples.x; x++)
            {
               if ((currentSamplingPoint.x - spherePosition.x) * (currentSamplingPoint.x - spherePosition.x) +
                      (currentSamplingPoint.y - spherePosition.y) * (currentSamplingPoint.y - spherePosition.y) +
                      (currentSamplingPoint.z - spherePosition.z) * (currentSamplingPoint.z - spherePosition.z) <=
                   sphereRadius * sphereRadius)
               {
                  field.get().overlapFractions[field.get().index] += 1.0;
               }
               currentSamplingPoint.x += sampleDistance.x;
            }
            currentSamplingPoint.y += sampleDistance.y;
         }
         currentSamplingPoint.z += sampleDistance.z;
      }

      field.get().overlapFractions[field.get().index] *= 1.0 / (nSamples.x * nSamples.y * nSamples.z);
      if (field.get().overlapFractions[field.get().index] > 0)
      {
         field.get().uids[field.get().index] = uid;
         field.get().index += 1;
      }
      assert(field.get().index < MaxParticlesPerCell);
   }
}*/

// functions to calculate Bs
template< int Weighting_T >
__device__ void calculateWeighting(real_t* weighting, const real_t& /*epsilon*/, const real_t& /*tau*/)
{
   WALBERLA_STATIC_ASSERT(Weighting_T == 1 || Weighting_T == 2);
}
template<>
__device__ void calculateWeighting< 1 >(real_t* weighting, const real_t& epsilon, const real_t& /*tau*/)
{
   *weighting = epsilon;
}
template<>
__device__ void calculateWeighting< 2 >(real_t* weighting, const real_t& epsilon, const real_t& tau)
{
   *weighting = epsilon * (tau - real_t(0.5)) / ((real_t(1) - epsilon) + (tau - real_t(0.5)));
}

__global__ void resetKernelSoA(walberla::cuda::FieldAccessor< uint_t > indicesField,
                               walberla::cuda::FieldAccessor< real_t > overlapFractionsField,
                               walberla::cuda::FieldAccessor< id_t > uidsField,
                               walberla::cuda::FieldAccessor< real_t > bnField)
{
   indicesField.set(blockIdx, threadIdx);
   overlapFractionsField.set(blockIdx, threadIdx);
   uidsField.set(blockIdx, threadIdx);
   bnField.set(blockIdx, threadIdx);

   for (uint i = 0; i < MaxParticlesPerCell; i++)
   {
      overlapFractionsField.get(i) = 0.0;
      uidsField.get(i)             = id_t(0);
   }
   indicesField.get() = 0;
   bnField.get()      = 0.0;
}

// TODO: look for better mapping method
template< int Weighting_T >
__global__ void particleAndVolumeFractionMappingKernelSoA(walberla::cuda::FieldAccessor< uint_t > indicesField,
                                                          walberla::cuda::FieldAccessor< real_t > overlapFractionsField,
                                                          walberla::cuda::FieldAccessor< id_t > uidsField,
                                                          walberla::cuda::FieldAccessor< real_t > bnField, real_t omega,
                                                          double3 spherePosition, real_t sphereRadius,
                                                          double3 blockStart, double3 dx, int3 nSamples, id_t uid)
{
   indicesField.set(blockIdx, threadIdx);
   overlapFractionsField.set(blockIdx, threadIdx);
   uidsField.set(blockIdx, threadIdx);
   bnField.set(blockIdx, threadIdx);

   double3 sampleDistance       = { 1.0 / (nSamples.x + 1) * dx.x, 1.0 / (nSamples.y + 1) * dx.y,
                                    1.0 / (nSamples.z + 1) * dx.z };
   double3 startSamplingPoint   = { (blockStart.x + threadIdx.x * dx.x + sampleDistance.x),
                                    (blockStart.y + blockIdx.x * dx.y + sampleDistance.y),
                                    (blockStart.z + blockIdx.y * dx.z + sampleDistance.z) };
   double3 currentSamplingPoint = startSamplingPoint;

   double3 minCornerSphere = { spherePosition.x - sphereRadius, spherePosition.y - sphereRadius,
                               spherePosition.z - sphereRadius };
   double3 maxCornerSphere = { spherePosition.x + sphereRadius, spherePosition.y + sphereRadius,
                               spherePosition.z + sphereRadius };

   if (startSamplingPoint.x + dx.x > minCornerSphere.x && startSamplingPoint.x < maxCornerSphere.x &&
       startSamplingPoint.y + dx.y > minCornerSphere.y && startSamplingPoint.y < maxCornerSphere.y &&
       startSamplingPoint.z + dx.z > minCornerSphere.z && startSamplingPoint.z < maxCornerSphere.z)
   {
      for (uint_t z = 0; z < nSamples.z; z++)
      {
         currentSamplingPoint.y = startSamplingPoint.y;
         for (uint_t y = 0; y < nSamples.y; y++)
         {
            currentSamplingPoint.x = startSamplingPoint.x;
            for (uint_t x = 0; x < nSamples.x; x++)
            {
               if ((currentSamplingPoint.x - spherePosition.x) * (currentSamplingPoint.x - spherePosition.x) +
                      (currentSamplingPoint.y - spherePosition.y) * (currentSamplingPoint.y - spherePosition.y) +
                      (currentSamplingPoint.z - spherePosition.z) * (currentSamplingPoint.z - spherePosition.z) <=
                   sphereRadius * sphereRadius)
               {
                  overlapFractionsField.get(indicesField.get()) += 1.0;
               }
               currentSamplingPoint.x += sampleDistance.x;
            }
            currentSamplingPoint.y += sampleDistance.y;
         }
         currentSamplingPoint.z += sampleDistance.z;
      }

      overlapFractionsField.get(indicesField.get()) *= 1.0 / (nSamples.x * nSamples.y * nSamples.z);
      calculateWeighting< Weighting_T >(&overlapFractionsField.get(indicesField.get()),
                                        overlapFractionsField.get(indicesField.get()), real_t(1.0) / omega);
      if (overlapFractionsField.get(indicesField.get()) > 0)
      {
         uidsField.get(indicesField.get()) = uid;
         indicesField.get() += 1;
         bnField.get() += overlapFractionsField.get(indicesField.get());
      }
      assert(indicesField.get() < MaxParticlesPerCell);
   }
}

// TODO: find better solution for template kernels
auto instance_with_weighting_1 = particleAndVolumeFractionMappingKernelSoA< 1 >;
auto instance_with_weighting_2 = particleAndVolumeFractionMappingKernelSoA< 2 >;

} // namespace cuda
} // namespace psm
} // namespace lbm_mesapd_coupling
} // namespace walberla
